#include "hip/hip_runtime.h"
/**
 * \Created on: Mar 18, 2020
 * \Author: Asena Durukan, Asli Altiparmak, Elif Ozbay, Hasan Ozan Sogukpinar, Nuri Furkan Pala
 * \file mplib.cu
 * \brief Implementation of mplib.h library.
 */

#include <stdio.h>
#include <stdlib.h>
#include <gmp.h>
#include "mplib.h"

void big_rand(ui z, ui_t l) {
	int i;

	for(i = 0; i < l; i++) {
		z[i] = ((ui_t)rand()) * ((ui_t)rand()) * ((ui_t)rand()) * ((ui_t)rand());
	}
}

void big_mod_rand(ui z, ui_t l, ui n, ui_t nl, ui mu, ui_t mul) {
	ui_t z_[2 * nl];
	int i;

	for(i = 0; i < l; i++) {
		z_[i] = ((ui_t)rand()) * ((ui_t)rand()) * ((ui_t)rand()) * ((ui_t)rand());
	}
	for(i = l; i < 2 * nl; i++) {
		z_[i] = 0L;
	}
	barret_reduction(z, z_, 2 * nl, n, nl, mu, mul);
}

void big_print(FILE *fp, ui a, ui_t al, char *s, char *R) {
    if(R != NULL) {
	    fprintf(fp, "%s := %s!(", s, R);
    } else {
        fprintf(fp, "%s := (", s);
    }
    fprintf(fp, "%u", a[0]);
    for(int i = 1; i < al; i++) {
        fprintf(fp, " + %u * (2^%d)^%d", a[i], W, i);
    }
    fprintf(fp, ");\n\n");
}

void big_is_equal(int *z, ui a, ui b, ui_t l) {
	int i;
	*z = 1;
	for(i = 0; i < l; i++) {
		if(a[i] != b[i]) {
			*z = 0;
		}
	}
}

void big_is_equal_ui(int *z, ui a, ui_t al, ui_t b) {
	int i;
	*z = 1;
	if(a[0] != b) {
		*z = 0;
	}
	for(i = 1; i < al; i++) {
		if(a[i] != 0) {
			*z = 0;
		}
	}
}

void big_add(ui z, ui a, ui_t al, ui b, ui_t bl) {
	int i;
	ui_t carry_bit = 0;

	for(i = 0; i < al; i++) {
		z[i] = a[i] + b[i] + carry_bit;
		if(z[i] < a[i]) {
			carry_bit = 1;
		} else if(z[i] > a[i]) {
			carry_bit = 0;
		}
	}
}

void big_mod_add(ui z, ui a, ui_t al, ui b, ui_t bl, ui n, ui_t nl, ui mu, ui_t mul) {
	int i;
	ui_t z_[2 * nl], carry_bit = 0;

	for(i = al + 1; i < 2 * nl; i++) {
		z_[i] = 0L;
	}
	for(i = 0; i < al; i++) {
		z_[i] = a[i] + b[i] + carry_bit;
		if(z_[i] < a[i]) {
			carry_bit = 1;
		} else if(z_[i] > a[i]) {
			carry_bit = 0;
		}
	}
	z_[al] = carry_bit;
	barret_reduction(z, z_, 2 * nl, n, nl, mu, mul);
}

void big_sub(ui z, int *d, ui a, ui_t al, ui b, ui_t bl) {
	int i;
	ui_t borrow_bit = 0;

	for(i = 0; i < al; i++) {
		z[i] = a[i] - b[i] - borrow_bit;
		if(z[i] < a[i]) {
			borrow_bit = 0;
		} else if(z[i] > a[i]) {
			borrow_bit = 1;
		}
	}
	*d = borrow_bit;
}

void big_mod_sub(ui z, ui a, ui_t al, ui b, ui_t bl, ui n, ui_t nl) {
	int i;
	ui_t z_[nl], borrow_bit = 0;

	for(i = 0; i < al; i++) {
		z_[i] = a[i] - b[i] - borrow_bit;
		if(z_[i] < a[i]) {
			borrow_bit = 0;
		} else if(z_[i] > a[i]) {
			borrow_bit = 1;
		}
	}
	if(borrow_bit) {
    	big_add(z, z_, nl, n, nl);
	} else {
		big_cpy(z, z_, 0, nl);
	}
}

void big_mul(ui z, ui a, ui_t al, ui b, ui_t bl) {
	int i, j;
	ui_t u, v;
	uni_t uv;

	for(i = 0; i <= al; i++) {
		z[i] = 0;
	}
 	for(i = 0; i < al; i++) {
	 	u = 0;
		for(j = 0; j < bl; j++) {
			uv = (uni_t)z[i + j] + (uni_t)a[i] * (uni_t)b[j] + (uni_t)u;
			u = uv >> W;
			v = uv & 0xFFFFFFFF;  // TODO: W != 32?
			z[i + j] = v;
		}
		z[i + bl] = u;
	}
}

void big_mod_mul(ui z, ui a, ui_t al, ui b, ui_t bl, ui n, ui_t nl, ui mu, ui_t mul) {
	int i, j;
	ui_t u, v, z_[2 * nl];
	uni_t uv;

	for(i = al + bl; i < 2 * nl; i++) {
		z_[i] = 0;
	}
	for(i = 0; i <= al; i++) {
		z_[i] = 0;
	}
 	for(i = 0; i < al; i++) {
	 	u = 0;
		for(j = 0; j < bl; j++) {
			uv = (uni_t)z_[i + j] + (uni_t)a[i] * (uni_t)b[j] + (uni_t)u;
			u = uv >> W;
			v = uv & 0xFFFFFFFF; // TODO: W != 32?
			z_[i + j] = v;
		}
		z_[i + bl] = u;
	}
	barret_reduction(z, z_, 2 * nl, n, nl, mu, mul);
}

void big_get_mu(ui mu, ui n, ui_t nl) {
	mpz_t mp_n, mp_b2k, mp_mu;

    mpz_init(mp_n);
    mpz_init(mp_b2k);
    mpz_init(mp_mu);

	mpz_set_ui(mp_b2k, 0L);
	mpz_set_ui(mp_mu, 0L);

	mpz_import(mp_n, nl, -1, 4, 0, 0, n);
	mpz_add_ui(mp_b2k, mp_b2k, 1);
	mpz_mul_2exp(mp_b2k, mp_b2k, W * (2 * nl));
	mpz_fdiv_q(mp_mu, mp_b2k, mp_n);
	mpz_export(mu, NULL, -1, 4, 0, 0, mp_mu);
}

void big_get_A24(ui z, ui A, ui n, ui_t nl, ui mu, ui_t mul, int *flag) {
	ui_t c_2[nl], c_4[nl], A2[nl], ic_4[nl];
	int i, ret;

	c_2[0] = 2L;
	c_4[0] = 4L;
	for (i = 1; i < nl; i++) {
		c_2[i] = 0L;
		c_4[i] = 0L;
	}
	big_mod_add(A2, A, nl, c_2, nl, n, nl, mu, mul);
	ret = big_invert(ic_4, c_4, nl, n, nl);
	if(ret) { // Inverse exists
		big_mod_mul(z, A2, nl, ic_4, nl, n, nl, mu, mul);
		*flag = 1;
	} else { // Inverse does not exist
		big_gcd(z, nl, c_4, nl, n, nl);
		*flag = 0;
	}
}

uni_t barret_reduction_UL(uni_t p, uni_t b, uni_t k, uni_t z, uni_t m, uni_t L) { // Calculate z mod p where z < 2^W and p < 2^W
    uni_t bkpp = (k + 1) * L;
    uni_t bkmp = (k - 1) * L;
    uni_t bkp = 1 << bkpp;
    uni_t bkm = 1 << bkmp;

    uni_t q = ((z >> bkmp) * m) >> bkpp;
    uni_t r = (z & (bkp - 1)) - ((q * p) & (bkp - 1));
    if(r < 0) {
        r = r + bkp;
    }
    while(r >= p) {
        r -= p;
    }

    return r;
}

// ml = 2 * nl
void barret_reduction(ui z, ui m, ui_t ml, ui n, ui_t nl, ui mu, ui_t mul) { // Calculate m mod n
    ui_t k = nl, md[k + 1], mdmu[mul + k + 1], q[mul], mm[k + 1], qn[mul + nl], qnm[k + 1], r2[k + 1], r3[k + 1];
    int i, b;

    big_cpy(md, m, k - 1, k + 1); // md = m / b^(k - 1)
    big_mul(mdmu, md, k + 1, mu, mul); // mdmu = md * mu
    big_cpy(q, mdmu, k + 1, mul); // q = (m / b^(k - 1) * mu) / b^(k + 1)
    big_cpy(mm, m, 0, k + 1); // mm = m mod b^(k + 1)
    big_mul(qn, q, mul, n, nl); // qn = q * n
    big_cpy(qnm, qn, 0, k + 1); // qnm = qn mod b^(k + 1)
    big_sub(r3, &i, mm, k + 1, qnm, k + 1); // r3 = mm - qnm
	big_cpy(z, r3, 0, k);
    big_sub(r2, &b, r3, nl, n, nl); // while r >= n do: r <- r - n
	r2[nl] = r3[nl] - b;
    while(!(r2[nl] >> (W - 1))) {
        big_cpy(z, r2, 0, k);
		big_cpy(r3, r2, 0, k + 1);
        big_sub(r2, &b, r3, nl, n, nl);
		r2[nl] = r3[nl] - b;
    }
}

// Using GMP for now
void big_gcd(ui d, ui_t dl, ui a, ui_t al, ui b, ui_t bl) {
    mpz_t mp_a, mp_b, mp_d;
	int i;

	for(i = 0; i < dl; i++) {
		d[i] = 0L;
	}
	mpz_init(mp_a);
    mpz_init(mp_b);
    mpz_init(mp_d);

    mpz_set_ui(mp_a, 0L);
	mpz_set_ui(mp_b, 0L);
    mpz_set_ui(mp_d, 0L);

	mpz_import(mp_a, al, -1, 4, 0, 0, a);
    mpz_import(mp_b, bl, -1, 4, 0, 0, b);
	mpz_gcd(mp_d, mp_a, mp_b);
	mpz_export(d, NULL, -1, 4, 0, 0, mp_d);
}

int big_invert(ui z, ui a, ui_t al, ui b, ui_t bl) {
	int i, ret;
	mpz_t mp_z, mp_a, mp_b;

	mpz_init(mp_z);
	mpz_init(mp_a);
	mpz_init(mp_b);

	mpz_import(mp_a, al, -1, 4, 0, 0, a);
	mpz_import(mp_b, bl, -1, 4, 0, 0, b);
	mpz_set_ui(mp_z, 0L);

	for(i = 0; i < bl; i++) {
		z[i] = 0L;
	}
	ret = mpz_invert(mp_z, mp_a, mp_b);
	mpz_export(z, NULL, -1, 4, 0, 0, mp_z);       // iY2Z = Inv(Y^2Z)

	return ret;
}

void memoryAllocationGPU (ui *deviceArray, ui_t arraySize) {
	//Memory Allocation for GPU
	hipMalloc(deviceArray, arraySize  * sizeof(ui_t));
	if (deviceArray == NULL)
		printf("deviceArray no space");
}

//__device__ void bigCpy(ui z, ui a, ui_t start, ui_t end, ui_t size) {
//	int i,j;
//	int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;
//	if(t_index < size){
//		int firstIndexZ = t_index * end, lastIndexZ = firstIndexZ + end;
//		int firstIndexA = t_index * start;
//		if(t_index == 0){
//			firstIndexA += start;
//		}
//		for(i = firstIndexZ, j = firstIndexA; i < lastIndexZ; i++, j++){
//			z[i] = a[j];
//		}
//	}
//}

__device__ void bigCpy(ui z, ui a, ui_t start, ui_t end, ui_t size) {
	int i,j;
	int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;
	if(t_index < size){
		int firstIndexZ = t_index * end, lastIndexZ = firstIndexZ + end;
		int firstIndexA = t_index * start;
		if(t_index == 0){
			firstIndexA += start;
		}
		bigCopy(z, firstIndexZ, lastIndexZ, a, firstIndexA);
	}
}

__device__ void bigMul(ui z, ui a, ui_t al, ui b, ui_t bl, ui_t size) {
	int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;
	int firstIndexA = t_index * al, lastIndexA = firstIndexA + al;
	ui_t u,low, high;
	if (t_index < size) {
		for(int i = firstIndexA; i <= lastIndexA; i++){
			z[i] = (ui_t)0;
		}
		for (int i = firstIndexA; i < lastIndexA; i++) {
			u = 0;
			for (int j = firstIndexA; j < lastIndexA; j++) {
				low = 0;
				high = 0;
				__mul_lo(low, a[i], b[j]);
				__mul_hi(high, a[i], b[j]);
				__add_cc(low, z[i+j], low);
				__addcy2(high);
				__add_cc(low, u, low);
				__addcy2(high);

				z[i+j] = low;
				u = high;
			}
			z[i + lastIndexA] = u;
		}
	}
}

__device__ void bigSub(ui z, ui controlBits, ui a, ui_t al, ui b , ui_t bl, ui_t size) {
	int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;
	int firstIndexA = t_index * al, lastIndexA = firstIndexA + al;
	int borrowbit = 0;

	if(t_index < size){
		for (int j = firstIndexA; j < lastIndexA; j++) {
			// c[j] = a[j] - b[j] - borrowBit
			__sub_cc(z[j], a[j], b[j]);
			__sub_cc(z[j], z[j], borrowbit);
			if(z[j]>a[j]){
				borrowbit = 1;
			}else if (z[j] < a[j]){
				borrowbit = 0;
			}
		}
		//__subcy(controlBits[t_index]);  //TODO : Add Macro
		controlBits[t_index] = borrowbit;
	}
}

__device__ float gpuGenerate(hiprandState* globalState) {
	int t_index = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState localState = globalState[t_index];
	float RANDOM = hiprand_uniform(&localState);
	globalState[t_index] = localState;
	return RANDOM;
}


__global__ void bigModRand(ui z, ui_t l, ui n, ui_t nl, ui mu, ui_t mul, ui_t size, hiprandState* globalState, ui_t seed){
	int t_index = threadIdx.x + blockIdx.x * blockDim.x;
	int i;

	if(t_index < size){
		ui z_ = new ui_t[2 * nl];
		int firstIndexZ_ = t_index * 2 * nl, lastIndexZ_ = firstIndexZ_ + (2 * nl);
		int halfIndexZ_ = ((lastIndexZ_ - firstIndexZ_ ) / 2 ) + firstIndexZ_;
		hiprand_init(seed, t_index, 0, &globalState[t_index]);
		for(i = firstIndexZ_; i < halfIndexZ_; i++) {
			z_[i] = (ui_t)(gpuGenerate(globalState) *  4294967295);
		}
		for(i = halfIndexZ_; i < lastIndexZ_; i++) {
			z_[i] = 0L;
		}
		barretReduction(z, z_, 2 * nl, n, nl, mu, mul,size);
	}
}

__global__ void bigModMul(ui z, ui a, ui_t al, ui b, ui_t bl, ui n, ui_t nl, ui mu, ui_t mul, ui_t size) {
	int i, j;
	ui z_ = new ui_t[2 * nl];
	ui_t u, low, high;
	int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;
	int firstIndexA = t_index * al, lastIndexA = firstIndexA + al;
	int firstIndexZ = t_index * 2 * nl, lastIndexZ = firstIndexZ + (2 * nl);
	if (t_index < size) {
		for(i = firstIndexZ; i < lastIndexZ; i++){
			z_[i] = 0;
		}
		for (i = firstIndexA; i < lastIndexA; i++) {
			u = 0;
			for (j = firstIndexA; j < lastIndexA; j++) {

				low = 0;
				high = 0;
				__mul_lo(low, a[i], b[j]);
				__mul_hi(high, a[i], b[j]);
				__add_cc(low, z_[i+j], low);
				__addcy2(high);
				__add_cc(low, u, low);
				__addcy2(high);

				z_[i + j] = low;
				u = high;
			}
			z_[i + lastIndexA] = u;
		}
		barretReduction(z, z_, 2 * nl, n, nl, mu, mul, size);
	}
}

__device__ void barretReduction(ui z, ui m, ui_t ml, ui n,ui_t nl, ui mu,ui_t mul, ui_t size){
    ui_t k = nl;
    ui md = new ui_t[k+1];
    ui mdmu = new ui_t[mul + k + 1];
    ui q = new ui_t[mul];
    ui mm = new ui_t[k + 1];
    ui qn = new ui_t[mul + nl];
    ui qnm = new ui_t[k + 1];
    ui r2 = new ui_t[k + 1];
    ui r3 = new ui_t[k + 1];
    //int i, j, b;
    ui controlBits = new ui_t[size];
    ui controlBitsi = new ui_t[size];
    int t_index = (blockDim.x * blockIdx.x) + threadIdx.x;

    bigCpy(md, m, k - 1, k + 1,size); // md = m / b^(k - 1)
    bigMul(mdmu, md, k + 1, mu, mul,size); // mdmu = md * mu
    bigCpy(q, mdmu, k + 1, mul,size); // q = (m / b^(k - 1) * mu) / b^(k + 1)
    bigCpy(mm, m, 0, k + 1,size); // mm = m mod b^(k + 1)
    bigMul(qn, q, mul, n, nl,size); // qn = q * n
    bigCpy(qnm, qn, 0, k + 1,size); // qnm = qn mod b^(k + 1)
    bigSub(r3, controlBitsi, mm, k + 1, qnm, k + 1,size); // r3 = mm - qnm
    bigCpy(z, r3, 0, k,size);
    bigSub(r2, controlBits, r3, nl, n, nl,size); // while r >= n do: r <- r - n
	r2[nl] = r3[nl] - controlBits[t_index];
    while(!(r2[nl] >> (W - 1))) {
    	bigCpy(z, r2, 0, k,size);
    	bigCpy(r3, r2, 0, k + 1,size);
        bigSub(r2, controlBits, r3, nl, n, nl,size);
		r2[nl] = r3[nl] - controlBits[t_index];
    }
}
